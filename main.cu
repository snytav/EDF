#include "hip/hip_runtime.h"
#include "maxwell.h"
#include "load_data.h"
#include "particle.h"

#include <string>
#include <vector>

#define N 200

double f[N];

double get_maxv(std::vector<Particle> & p)
{
    double maxv = 0.0;
    
    for(int j = 0;j < p.size();j++)
    {
        double pu,pv,pw,ps,v;//,hv = 0.02/num;
	int n;
	
	pu = p[j].pu;
	pv = p[j].pv;
	pw = p[j].pw;
	
	ps = pu * pu + pv * pv + pw * pw;
	ps = pow(((pu * pu + pv * pv + pw * pw) + 1.0),-0.5);
	
	
	
	v = pu*ps;
	if(fabs(v) > maxv) maxv = fabs(v);
    }
    
    return maxv;
}

void getEDF(std::vector<Particle> & p,double *f,int num,double q,int nt)
{
    static int first  = 1;
    double maxv,hv;
//     e hv = 2*maxv/num;
    
    if(first == 1)
    {
       for(int i = 0;i < N;i++)
       {
	   f[i] = 0.0; 
	   
       }
      
       
       first  = 0;
    }
    maxv = get_maxv(p);
    hv = 2*maxv/num;
    
    FILE *f1;
    char fname[200];
    
    sprintf(fname,"electrons%05d_%10.3e.txt",nt,q);
    
    if((f1 = fopen(fname,"wt")) == NULL) return;
    
    
    
    for(int j = 0;j < p.size();j++)
    {
        double pu,pv,pw,ps,v;
	int n;
	
	pu = p[j].pu;
	pv = p[j].pv;
	pw = p[j].pw;
	
	ps = pu * pu + pv * pv + pw * pw;
	ps = pow(((pu * pu + pv * pv + pw * pw) + 1.0),-0.5);
	
	
	
	v = pu*ps;

	n = v/hv+num/2;
	f[n] += q;
	
	fprintf(f1,"%10d x %15.5e px %15.5e vx %15.5e n %5d \n",j,p[j].x,p[j].pu,v,n);
//     
    }
    fclose(f1);
    for(int i = 0;i < num;i++)
       {
	   f[i] /= p.size(); 
	   
       }
    sprintf(fname,"edf%05d.txt",nt,q);
    
    if((f1 = fopen(fname,"wt")) == NULL) return;  
    for(int i = 0;i < num;i++)
       {
	   fprintf(f1,"%5d %15.5e %15.5e\n",i,(i-num/2)*hv,f[i]); 
	   
       }   
       fclose(f1);
}


int main(int argc,char *argv[])
{
    double q = 1.0,maxv;  
    int    nt  = 128;//atoi(argv[1]);
    
  
    std::vector<Particle> ion_vp,el_vp,beam_vp;
    
    LoadParticleData(nt,ion_vp,el_vp,beam_vp,100,4,4);
    
    std::vector<Particle> all_el;
    all_el.reserve( el_vp.size() + beam_vp.size() );                // preallocate memory
    all_el.insert( all_el.end(), el_vp.begin(), el_vp.end() );        // add A;
    all_el.insert( all_el.end(), beam_vp.begin(), beam_vp.end() );        // add B;
    
    maxv = get_maxv(all_el);
    
    getEDF(all_el,f,N,q,nt);
//     q = 1.0/2000.0;
//     getEDF(beam_vp,f,N,q,nt);
//     FILE *f;
//     char fname[200];
//     
//     sprintf(fname,"electrons%05d.txt",1);
//     
//     if((f = fopen(fname,"wt")))
//     {
//        for(int j = 0;j < p.size();j++)
//        {
// 	   fprintf(f,"%10d x %15.5e px %15.5e vx %15.5e \n",j,el_vp[j].pu,);
//        }

    return 0;
}
